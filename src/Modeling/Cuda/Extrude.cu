#include "Extrude.h"
#include "Topology/PointSet.h"
#include "GLSurfaceVisualModule.h"
#include "GLWireframeVisualModule.h"
#include "GLPointVisualModule.h"
#include "EarClipper.h"

namespace dyno
{
	template<typename TDataType>
	ExtrudeModel<TDataType>::ExtrudeModel()
		: ParametricModel<TDataType>()
	{
		this->varHeight()->setRange(0.001f, 10.0f);

		this->stateTriangleSet()->setDataPtr(std::make_shared<TriangleSet<TDataType>>());

		auto glModule = std::make_shared<GLSurfaceVisualModule>();
		glModule->setColor(Color(0.8f, 0.52f, 0.25f));
		glModule->setVisible(true);
		this->stateTriangleSet()->connect(glModule->inTriangleSet());
		this->graphicsPipeline()->pushModule(glModule);

		auto glModule2 = std::make_shared<GLPointVisualModule>();
		glModule2->setColor(Color(1.0f, 1.0f, 1.0f));
		glModule2->varPointSize()->setValue(0.01);
		this->stateTriangleSet()->connect(glModule2->inPointSet());
		this->graphicsPipeline()->pushModule(glModule2);

		auto glModule3 = std::make_shared<GLWireframeVisualModule>();
		glModule3->setColor(Color(1.0f, 1.0f, 1.0f));
		this->stateTriangleSet()->connect(glModule3->inEdgeSet());
		this->graphicsPipeline()->pushModule(glModule3);

		auto callback = std::make_shared<FCallBackFunc>(std::bind(&ExtrudeModel<TDataType>::varChanged, this));

		this->varLocation()->attach(callback);
		this->varScale()->attach(callback);
		this->varRotation()->attach(callback);

		this->varRow()->attach(callback);
		this->varHeight()->attach(callback);
		this->varReverseNormal()->attach(callback);
		this->varCurve()->attach(callback);


	}
	template<typename TDataType>
	void ExtrudeModel<TDataType>::resetStates()
	{
		varChanged();
	}

	template<typename TDataType>
	void ExtrudeModel<TDataType>::varChanged()
	{
		auto center = this->varLocation()->getData();
		auto rot = this->varRotation()->getData();
		auto scale = this->varScale()->getData();

		enum PointMode
		{
			UseInput = 0,
			UseCurve = 1,
		};

		PointMode mPointMode = PointMode::UseCurve;

		int columns_i = 0;
		if (!this->inPointSet()->isEmpty()) 
		{
			columns_i = this->inPointSet()->getData().getPointSize();
			mPointMode = PointMode::UseInput;
		}
		else if (this->varCurve()->getValue().getPointSize()) 
		{
			columns_i = this->varCurve()->getValue().getPointSize();
			mPointMode = PointMode::UseCurve;
		}
		else
		{
			columns_i = 0;
		}
		
		if (columns_i >= 3)
		{
		//�����ǲ����Ĺ���

			Real HeightValue = this->varHeight()->getData();
			Real RowValue = this->varRow()->getData();
			Real tempRow = 0;

			std::vector<Coord> vertices;
			CArray<Coord> capPoint;

			for (int i = 0; i <= RowValue; i++) 
			{
				Real tempy = HeightValue * i / RowValue;
				Vec3f position;

				if (mPointMode == PointMode::UseInput)
				{
					if (i == 0)
					{
						DArray<Coord> sa = this->inPointSet()->getData().getPoints();
						capPoint.assign(sa);
					}

					for (int k = 0; k < columns_i; k++)
					{
						position = { capPoint[k][0] , capPoint[k][1] + tempy ,capPoint[k][2] };

						vertices.push_back(position);

					}
				}
				else if (mPointMode == PointMode::UseCurve)
				{
					for (int k = 0; k < columns_i; k++)
					{
						auto curvePoint = this->varCurve()->getValue().getPoints();

						position = { float(curvePoint[k].x) , float(tempy) ,float(curvePoint[k].y) };
						vertices.push_back(position);

						if (i == 0 )
							capPoint.pushBack(position);

					}

				}
			}




			//�����ǵײ����ϲ���Ĺ���
			std::vector<TopologyModule::Triangle> triangle;

			int pt_side_len = vertices.size();

			//�����ǲ���Ĺ���
			for (int rowl = 0; rowl <= RowValue - 1; rowl++)
			{

				for (int faceid = 0; faceid < columns_i; faceid++)
				{
					if (faceid != columns_i - 1)
					{
						triangle.push_back(TopologyModule::Triangle(columns_i + faceid + rowl * columns_i, 0 + faceid + rowl * columns_i, 1 + faceid + rowl * columns_i));
						triangle.push_back(TopologyModule::Triangle(columns_i + 1 + faceid + rowl * columns_i, columns_i + faceid + rowl * columns_i, 1 + faceid + rowl * columns_i));
					}
					else
					{
						triangle.push_back(TopologyModule::Triangle(1 + 2 * faceid + rowl * columns_i, 0 + faceid + rowl * columns_i, 0 + rowl * columns_i));
						triangle.push_back(TopologyModule::Triangle(1 + faceid + rowl * columns_i, 1 + 2 * faceid + rowl * columns_i, 0 + rowl * columns_i));
					}

				}
			}

			int pt_len = vertices.size() - 2;
			int top_pt_len = vertices.size() - 2 - pt_side_len;
			int addnum = 0;



			//transform


			Quat<Real> q = computeQuaternion();

			q.normalize();

			auto RV = [&](const Coord& v)->Coord {
				return center + q.rotate(v - center);
			};

			int numpt = vertices.size();

			for (int i = 0; i < numpt; i++)
			{
				//vertices[i][1] -= height / 2;
				vertices[i] = RV(vertices[i] * scale + RV(center));
			}



			if (this->varReverseNormal()->getData() == true)
			{
				int trinum = triangle.size();
				for (int i = 0; i < trinum; i++)
				{
					int temp;
					temp = triangle[i][0];
					triangle[i][0] = triangle[i][2];
					triangle[i][2] = temp;
				}
			}


			EarClipper<DataType3f> sab;
			std::vector<TopologyModule::Triangle> triangleCap;

			sab.polyClip(capPoint, triangleCap);
			int addnum2 = vertices.size() - capPoint.size();

			
			for (int i = 0; i < triangleCap.size(); i++)
			{
				triangle.push_back(triangleCap[i]);
				triangle.push_back(TopologyModule::Triangle(triangleCap[i][0] + addnum2, triangleCap[i][1] + addnum2, triangleCap[i][2] + addnum2));
			}

			auto triangleSet = this->stateTriangleSet()->getDataPtr();

			triangleSet->setPoints(vertices);
			triangleSet->setTriangles(triangle);

			triangleSet->update();

			vertices.clear();
			triangle.clear();
		}
		
	}


	DEFINE_CLASS(ExtrudeModel);
}